#include "hip/hip_runtime.h"
// Project Euler problem 1
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void filterKernel(int *array)
{
	int i = threadIdx.x;
	// determine module of thread index value
	if ((i % 3 == 0 || i % 5 == 0 ) && i < 1000)
	{
		array[i] = i;
	}
	else
	{
		array[i] = 0;
	}

}

__global__ void reduceKernel(int *array)
{
	int i = threadIdx.x;

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (i < s)
		{
			array[i] += array[i + s];
		}
		__syncthreads();
	}
}


int main()
{
	// task: Find the sum of all the multiples of 3 or 5 below 1000.
	const int task_size = 1024; // upper bound of task is hard-coded in kernel
	
	// inputs
	int *d_array;
	hipMalloc((void**)&d_array, sizeof(int) * task_size);

	// run filter kernel
	filterKernel <<<1, task_size>>>(d_array);

	// Check for any errors launching the kernel
	{
		hipError_t cudaStatus;
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	}

	// run reduce kernel
	reduceKernel << <1, task_size >> >(d_array);

	// Check for any errors launching the kernel
	{
		hipError_t cudaStatus;
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
	}

	// retrieve result
	int result;
	hipMemcpy(&result, d_array, sizeof(int), hipMemcpyDeviceToHost);

	printf("The result is %d\n", result);

	// clean up
Error:
	hipFree(d_array);
    return 0;
}

/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
